#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        std::cout << "hipGetDeviceCount returned error code " << error_id
                  << " (" << hipGetErrorString(error_id) << ")\n";
        return 1;
    }

    if (deviceCount == 0) {
        std::cout << "No CUDA-capable devices detected.\n";
        return 0;
    }

    std::cout << "Detected " << deviceCount << " CUDA-capable device(s).\n";

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout << "\nDevice " << dev << ": " << deviceProp.name << "\n";
        std::cout << "  Compute capability: " << deviceProp.major << "." << deviceProp.minor << "\n";
        std::cout << "  Total global memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB\n";
        std::cout << "  Multiprocessors: " << deviceProp.multiProcessorCount << "\n";
    }

    return 0;
}
